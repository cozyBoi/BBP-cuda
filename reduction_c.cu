
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10000
#define BLOCK_SIZE 16
__global__ void reduce0(int *g_idata, int *g_odata) {
    __shared__ int sdata[16];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) { 
        if (tid % (2*s) == 0) {
           sdata[tid] = sdata[tid + s] > sdata[tid] ? sdata[tid + s] : sdata[tid]; 
        }
        __syncthreads(); 
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
    srand(time(NULL));
    int a[N], b[N];
    
    
    for(int i = 0; i < N; i++){
        a[i] = rand() % 1000;
        //printf("%d ", a[i]);
    }
    //printf("\n");

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows, 1); //.x
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1); //.y

    
    int*d_a, *d_b;
    hipMalloc((void **)&d_a, N*sizeof(int));
    hipMalloc((void **)&d_b, N*sizeof(int));
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);

    reduce0<<<dimGrid, dimBlock>>> (d_a, d_b);
    
    hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);

    while(1){
        reduce0<<<dimGrid, dimBlock>>> (d_b, d_b);
    
        hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);

        if(b[1] == 0) break;
    }

    /*
    printf("a: ");
    for(int i = 0;i < N; i++){
        printf("%d ", a[i]);
    }
    printf("\n");*/

    /*
    printf("b: ");
    for(int i = 0;i < N; i++){
        printf("%d ", b[i]);
    }
    printf("\n");*/
    printf("max : %d\n", b[0]);
    return 0;
}