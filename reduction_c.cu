
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000

__global__ void reduce0(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    for (unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) { 
            sdata[index] = sdata[index + s] > sdata[index] ? sdata[index + s]  : sdata[index];
        }
        __syncthreads();
     }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
    dim3 dimGrid(8, 1, 1);
    dim3 dimBlock(8, 1, 1);
    int a[N], b[N];
    int*d_a, *d_b;
    hipMalloc((void **)&d_a, N*sizeof(int));
    hipMalloc((void **)&d_b, N*sizeof(int));
    hipMemcpy(d_a, &a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, N*sizeof(int), hipMemcpyHostToDevice);

    reduce0<<<dimGrid, dimBlock>>> (d_a, d_b);
    

    return 0;
}