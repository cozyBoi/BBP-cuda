#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define THREADS_PER_BLOCK 512
#define BLOCK_NUM 16
#define BLOCK_SIZE 16

__global__ void maxReduction(int *arr, int *res)
{
	__shared__ int tmp[10000];
	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	tmp[tid] = arr[idx];

  __syncthreads();
	for(int i = 1; i < blockDim.x; i *= 2) {
		if(tid % (2*i) == 0) {
			if(tmp[tid] < tmp[tid+i])
				tmp[tid] = tmp[tid+i];
		}
		__syncthreads();
	}
  
  if(tid == 0)
		res[blockIdx.x] = tmp[0];
}

int main(int argc, char *argv[])
{
    unsigned int size = atoi(argv[1]);

    int *arr, *res;
    arr = (int *) malloc(sizeof(int) * size);
    res = (int *) malloc(sizeof(int) * size);
    
    srand(time(NULL)); 
    for(int i = 0; i < size; i++)
      arr[i] = rand() % size;
  
    int *d_arr, *d_res, *d_tmp;
    hipMalloc((void**) &d_arr, size * sizeof(int));
    hipMalloc((void**) &d_res, size * sizeof(int));
    d_tmp = d_arr;

    hipMemcpy(d_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(BLOCK_NUM);
    
    float time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipEventRecord(start);  

    do {
      maxReduction<<<dimGrid, dimBlock>>> (d_tmp, d_res);
      hipMemcpy(res, d_res, size * sizeof(int), hipMemcpyDeviceToHost);
      d_tmp = d_res;
      size >>= 1;
    } while (res[1] != 0);

    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("Array Size: %d, Elapsed time : %.5f, Result : %d\n", size, time, res[0]);

    hipFree(d_arr);
    hipFree(d_res);
    free(arr);
    free(res);
    return 0;
}