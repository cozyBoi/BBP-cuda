
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10000
#define BLOCK_SIZE 16
__global__ void reduce0(int *g_idata, int *g_odata) {
    __shared__ int sdata[16];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    for (unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) { 
            sdata[index] = sdata[index + s] > sdata[index] ? sdata[index + s]  : sdata[index];
        }
        __syncthreads();
     }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int a[N], b[N];
int main() {
    srand(time(NULL));
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    for(int i = 0; i < N; i++){
        a[i] = rand() % 1000;
        //printf("%d ", a[i]);
    }
    //printf("\n");

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows, 1); //.x
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1); //.y

    
    int*d_a, *d_b;
    hipMalloc((void **)&d_a, N*sizeof(int));
    hipMalloc((void **)&d_b, N*sizeof(int));
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);


    hipEventRecord(start);
    reduce0<<<dimGrid, dimBlock>>> (d_a, d_b);
    hipEventRecord(stop);

    hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);

    while(1){
        reduce0<<<dimGrid, dimBlock>>> (d_b, d_b);
    
        hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);

        if(b[1] == 0) break;
    }
    
    /*
    printf("a: ");
    for(int i = 0;i < N; i++){
        printf("%d ", a[i]);
    }
    printf("\n");*/

    /*
    printf("b: ");
    for(int i = 0;i < N; i++){
        printf("%d ", b[i]);
    }
    printf("\n");*/
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time : %f\n", milliseconds);
    printf("max : %d\n", b[0]);
    return 0;
}