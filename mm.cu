
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 3
#define N_2 N*N

#define BLOCK_SIZE 16

__global__ void mm_kernel(float* A, float* B, double* C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < N && col < N) {
        double tmp = 0;
        for (int i = 0; i < N; ++i) {
             tmp += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = tmp;
    } 
}

int main() {
    srand(time(NULL));

    //dim3 dimGrid(3, 3, 1);
    //dim3 dimBlock(N/3, N/3, 1);
    float a[N_2], b[N_2];
    double c[N_2];

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i*N + j] = rand() % 10 + 1;
            b[i*N + j] = rand() % 10 + 1;
        }
    }
    /*
    printf("a:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", a[i*N + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");

    printf("b:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", b[i*N + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");*/

    float*d_a, *d_b;
    double *d_c;
    hipMalloc((void **)&d_a, N_2*sizeof(float));
    hipMalloc((void **)&d_b, N_2*sizeof(float));
    hipMalloc((void **)&d_c, N_2*sizeof(double));
    hipMemcpy(d_a, a, N_2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N_2*sizeof(float), hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows, 1); //.x
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1); //.y

    mm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c);
    
    hipMemcpy(c, d_c, N*N*sizeof(double), hipMemcpyDeviceToHost);
    /*
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%lf ", c[i*N + j]);
        }
        printf("\n");
    }*/
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}