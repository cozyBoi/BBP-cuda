
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 9
#define N_2 N*N

__global__ void mm_kernel(float* A, float* B, float* C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < N && col < N) {
        for (int i = 0; i < N; ++i) {
            C[row * N + col] += A[row * N + i] * B[i * N + col];
        }
        printf("%d %d %d", row, col, C[row * N + col]);
    } 
}

int main() {
    srand(time(NULL));

    dim3 dimGrid(3, 3, 1);
    dim3 dimBlock(N/3, N/3, 1);
    float a[N], b[N], C[N_2];

    for(int i = 0; i < N; i++){
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    float*d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N*sizeof(float));
    hipMalloc((void **)&d_b, N*sizeof(float));
    hipMalloc((void **)&d_c, N_2*sizeof(float));
    hipMemcpy(d_a, &a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, N*sizeof(float), hipMemcpyHostToDevice);

    mm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c);
    
    return 0;
}