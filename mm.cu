
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 3
#define N_2 N*N

__global__ void mm_kernel(float* A, float* B, float* C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < N && col < N) {
        for (int i = 0; i < N; ++i) {
            C[row * N + col] += A[row * N + i] * B[i * N + col];
        }
        printf("%d %d %d", row, col, C[row * N + col]);
    } 
}

int main() {
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);
    float a[N], b[N], C[N_2];
    float*d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N*sizeof(float));
    hipMalloc((void **)&d_b, N*sizeof(float));
    hipMalloc((void **)&d_c, N_2*sizeof(float));
    hipMemcpy(d_a, &a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, N*sizeof(float), hipMemcpyHostToDevice);

    mm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c);
    
    return 0;
}