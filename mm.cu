
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 3
#define N_2 N*N

__global__ void mm_kernel(float* A, float* B, float* C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < N && col < N) {
        for (int i = 0; i < N; ++i) {
            C[row * N + col] += A[row * N + i] * B[i * N + col];
        }
    } 
}

int main() {
    srand(time(NULL));

    //dim3 dimGrid(3, 3, 1);
    //dim3 dimBlock(N/3, N/3, 1);
    dim3 dimGrid(2, 1, 1);
    dim3 dimBlock(2, 1, 1);
    float a[N], b[N], c[N_2];

    for(int i = 0; i < N; i++){
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }
    for(int i = 0; i < N; i++){
        printf("%f ", a[i]);
    }
    printf("\n");

    for(int i = 0; i < N; i++){
        printf("%f ", b[i]);
    }
    printf("\n");

    float*d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N*sizeof(float));
    hipMalloc((void **)&d_b, N*sizeof(float));
    hipMalloc((void **)&d_c, N_2*sizeof(float));
    hipMemcpy(d_a, &a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, N*sizeof(float), hipMemcpyHostToDevice);

    mm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c);
    
    hipMemcpy(d_c, &c, N*N*sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", c[i*N + j]);
        }
        printf("\n");
    }

    return 0;
}