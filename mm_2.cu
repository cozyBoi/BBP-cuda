
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 3
#define N_2 N*N

#define BLOCK_SIZE 32

float a[N_2], b[N_2];
    float c[N_2];

__global__ void mm_kernel(float* A, float* B, float* C) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int thx = threadIdx.x, thy = threadIdx.y;
    unsigned int grdx = gridDim.x, grdy = gridDim.y;
    //const unsigned int bx = BLOCK_X, by = BLOCK_Y;
    const unsigned int bx = blockDim.x, by = blockDim.y;

    __shared__ float a[BLOCK_SIZE][BLOCK_SIZE], b[BLOCK_SIZE][BLOCK_SIZE];
    if (row < N && col < N) {
        float tmp = 0;
        for (int i = 0; i < N / BLOCK_SIZE; ++i) { //2. grd를 넘기면 멈춤
            a[thx][thy] = A[row*N+i*by+thx]; //1. block을 옮겨다님
            b[thy][thx] = B[col+N*(i*bx+thy)];
            __syncthreads(); 
            for (unsigned int j=0; j < bx; j++){
                tmp += a[j][thx]*b[j][thy];
            }
            __syncthreads(); 
            //tmp += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = tmp;
    } 
}

int main() {
    srand(time(NULL));

    //dim3 dimGrid(3, 3, 1);
    //dim3 dimBlock(N/3, N/3, 1);
    

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i*N + j] = rand() % 10 + 1;
            b[i*N + j] = rand() % 10 + 1;
        }
    }
    
    printf("a:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", a[i*N + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");

    printf("b:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", b[i*N + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    float*d_a, *d_b;
    float *d_c;
    hipMalloc((void **)&d_a, N_2*sizeof(float));
    hipMalloc((void **)&d_b, N_2*sizeof(float));
    hipMalloc((void **)&d_c, N_2*sizeof(float));
    hipMemcpy(d_a, a, N_2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N_2*sizeof(float), hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows, 1); //.x
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1); //.y

    hipEventRecord(start);
    mm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c);
    hipEventRecord(stop);

    hipMemcpy(c, d_c, N*N*sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time : %f\n", milliseconds);

    
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%lf ", c[i*N + j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}