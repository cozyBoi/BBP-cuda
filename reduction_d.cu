#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 10000

__global__ void reduce0(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // v                                                                            do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] = sdata[tid + s] >  sdata[tid] ? sdata[tid + s] : sdata[tid]; 
        }
        __syncthreads(); 
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
    dim3 dimGrid(8, 1, 1);
    dim3 dimBlock(8, 1, 1);
    int a[N], b[N], C[N_2];
    int*d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N*sizeof(int));
    hipMalloc((void **)&d_b, N*sizeof(int));
    hipMemcpy(d_a, &a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, N*sizeof(int), hipMemcpyHostToDevice);

    reduce0<<<dimGrid, dimBlock>>> (d_a, d_b);
    

    return 0;
}