
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 7000
#define N_2 N*N

#define BLOCK_SIZE 32

float a[N_2], b[N_2];
float c[N_2];
__global__ void mm_kernel(float* A, float* B, float* C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < N && col < N) {
        float tmp = 0;
        for (int i = 0; i < N; ++i) {
             tmp += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = tmp;
    } 
}

int main() {
    srand(time(NULL));

    //dim3 dimGrid(3, 3, 1);
    //dim3 dimBlock(N/3, N/3, 1);
    

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i*N + j] = rand() % 10 + 1;
            b[i*N + j] = rand() % 10 + 1;
        }
    }
    /*
    printf("a:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", a[i*N + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");

    printf("b:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%f ", b[i*N + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");*/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    float*d_a, *d_b;
    float *d_c;
    hipMalloc((void **)&d_a, N_2*sizeof(float));
    hipMalloc((void **)&d_b, N_2*sizeof(float));
    hipMalloc((void **)&d_c, N_2*sizeof(float));
    hipMemcpy(d_a, a, N_2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N_2*sizeof(float), hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows, 1); //.x
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1); //.y

    hipEventRecord(start);
    mm_kernel<<<dimGrid, dimBlock>>> (d_a, d_b, d_c);
    hipEventRecord(stop);

    hipMemcpy(c, d_c, N*N*sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time : %f\n", milliseconds);

    /*
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%lf ", c[i*N + j]);
        }
        printf("\n");
    }*/
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}