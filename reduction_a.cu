#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define N 10000

int arr[N];

int log2_N(int n){
    int a = 1;
    int i = 0;
    for(i = 0; a <= n; i++){}
        a *= 2;
    }
    return i;
}

int main() {
    srand(time(NULL));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    for(int i = 0; i < N; i++){
        arr[i] = rand() % 1000;
    }

    hipEventRecord(start);
    int maxVal = arr[0];
    //int log2_Nnum = log_2(N);
    for (unsigned int s=1; s < N; s *= 2) {
        for (unsigned int i=1; i < N; i++) {
            if (i + s < N) { 
                arr[i] = arr[i + s] > arr[i] ? arr[i + s]  : arr[i];
            }
        }
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time : %f\n", milliseconds);

    printf("%d\n", maxVal);
    return 0;
}