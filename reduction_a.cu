
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define N 10000

int main() {
    srand(time(NULL));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int arr[N];
    for(int i = 0; i < N; i++){
        arr[i] = rand() % 1000;
    }

    hipEventRecord(start);
    int maxVal = arr[0];
    for(int i = 0; i < N; i++){
        maxVal = maxVal < arr[i] ? arr[i] : maxVal;
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time : %f\n", milliseconds);

    printf("%d\n", maxVal);
    return 0;
}