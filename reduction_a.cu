
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define N 10000

int arr[N];

int main() {
    srand(time(NULL));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    for(int i = 0; i < N; i++){
        arr[i] = rand() % 1000;
    }

    hipEventRecord(start);
    
    for (unsigned int s=1; s < N; s *= 2) {
        for (unsigned int i=0; i < N; i++) {
            if (i + s < N) { 
                arr[i] = arr[i + s] > arr[i] ? arr[i + s]  : arr[i];
            }
        }
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time : %f\n", milliseconds);

    printf("%d\n", arr[0]);
    return 0;
}